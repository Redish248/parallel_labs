
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <sys/time.h>

using namespace std;

const int A = 936;

__global__ void map_m1(double* m1_v, int size) {
    //линейный индекс потока
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    //сколько один поток выполняет
    unsigned int threadsNum = blockDim.x * gridDim.x;
    for (unsigned int i = id; i < size; i += threadsNum) {
        m1_v[i] = cosh(m1_v[i]) + 1;
    }
}

__global__ void map_m2(double* m2_v, double* m2_copy_v, int size) {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int threadsNum = blockDim.x * gridDim.x;
    for (unsigned int i = id; i < size; i += threadsNum) {
        if (i == 0) {
            m2_v[i] = fabs((double) 1 / tan(m2_v[i]));
        } else {
            m2_v[i] = fabs((double) 1 / tan(m2_v[i] + m2_copy_v[i - 1]));
        }
    }
}

__global__ void merge(const double* m1_v, double* m2_v, int size) {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int threadsNum = blockDim.x * gridDim.x;
    for (unsigned int i = id; i < size; i += threadsNum) {
        m2_v[i] = (double) m1_v[i] / m2_v[i];
    }
}

/* comb_sort: function to find the new gap between the elements */
void comb_sort(double data[], int size) { //
    double factor = 1.2473309; // фактор уменьшения
    long step = size - 1; // шаг сортировки

    while (step >= 1) {
        for (int i = 0; i + step < size; i++) {
            if (data[i] > data[i + step]) {
                double tmp = data[i];
                data[i] = data[i + step];
                data[i + step] = tmp;
            }
        }
        step /= factor;
    }
}

// data - sorted array !
double reduce(double data[], int size) {
    double result = 0;

    int j = 0;
    while (j < size && data[j] == 0) {
        j++;
    }
    double min = data[j];

    for (int i = 0; i < size; i++) {
        if (((long) (data[i] / min) % 2) == 0) {
            result += sin(data[i]);
        }
    }

    return result;
}

int main(int argc, char *argv[]) {
    int N, K;
    struct timeval T1, T2;
    long delta_ms;

    if (argc < 2) {
        printf("Need to add size of array as input arguments\n");
        return -1;
    }

    N = atoi(argv[1]);
    if (argc >= 3) {
        K = atoi(argv[2]);
    } else K = 100;

    double *m1, *m2, *m2_copy;
    double *m1v, *m2v, *m2_copyv;

    m1 = (double *) malloc(N * sizeof(double));
    m2 = (double *) malloc(N / 2 * sizeof(double));
    m2_copy = (double *) malloc(N / 2 * sizeof(double));

    //выделение памяти на устройстве
    hipMalloc(&m1v, sizeof(double) * N);
    hipMalloc(&m2v, sizeof(double) * N / 2);
    hipMalloc(&m2_copyv, sizeof(double) * N / 2);

    gettimeofday(&T1, nullptr); //запомнить текущее время T1

    //для сихронизации потоков
    hipEvent_t syncEvent;
    hipEventCreate(&syncEvent);    //Создаем event
    hipEventRecord(syncEvent, nullptr);  //Записываем event

    //расчёт gridSize и blockSize для m1
    int gridSize, blockSize, minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, map_m1, 0, N);
    gridSize = (N + blockSize - 1) / blockSize;

    //расчёт gridSize и blockSize для m2
    int minGridSize2, blockSize2, gridSize2;
    hipOccupancyMaxPotentialBlockSize(&minGridSize2, &blockSize2, map_m2, 0, N / 2);
    gridSize2 = (N / 2  + blockSize2 - 1) / blockSize2;

    long delta_gen, delta_map, delta_merge, delta_sort, delta_reduce;
    struct timeval T0, T_generate, T_map, T_merge, T_sort, T_result;

    for (unsigned int ink = 0; ink < K; ink++) {
        gettimeofday(&T0, nullptr);

        //======================GENERATE======================
        unsigned int tmp1 = ink;
        unsigned int tmp2 = ink;
        //Заполнить массив исходных данных размером N
        for (int j = 0; j < N; j++) {
            double value = 1 + rand_r(&tmp1) % (A - 1);
            m1[j] = value;
        }

        for (int j = 0; j < N / 2; j++) {
            double value = A + rand_r(&tmp2) % (A * 10 - A);
            m2[j] = value;
            m2_copy[j] = value;
        }

        //копирование данных после инициализации
        hipMemcpy(m1v, m1, sizeof(double) * N, hipMemcpyHostToDevice);
        hipMemcpy(m2v, m2, sizeof(double) * N / 2, hipMemcpyHostToDevice);
        hipMemcpy(m2_copyv, m2_copy, sizeof(double) * N / 2, hipMemcpyHostToDevice);

        gettimeofday(&T_generate, nullptr);
        delta_gen = 1000 * (T_generate.tv_sec - T0.tv_sec) + (T_generate.tv_usec - T0.tv_usec) / 1000;




        //======================MAP======================

        map_m1<<<gridSize, blockSize>>>(m1v, N);
        map_m2<<<gridSize2, blockSize2>>>(m2v, m2_copyv, N / 2);

        //Хендл event'а
        hipEventSynchronize(syncEvent);  //Синхронизируем event

        hipMemcpy(m1, m1v, sizeof(double) * N, hipMemcpyDeviceToHost);
        hipMemcpy(m2, m2v, sizeof(double) * N / 2, hipMemcpyDeviceToHost);

        gettimeofday(&T_map, nullptr);
        delta_map = 1000 * (T_map.tv_sec - T_generate.tv_sec) + (T_map.tv_usec - T_generate.tv_usec) / 1000;




        //======================MERGE======================
        merge<<<gridSize2, blockSize2>>>(m1v, m2v, N / 2);

        hipEventSynchronize(syncEvent);  //Синхронизируем event
        hipMemcpy(m2, m2v, sizeof(double) * N / 2, hipMemcpyDeviceToHost);

        gettimeofday(&T_merge, nullptr);
        delta_merge = 1000 * (T_merge.tv_sec - T_map.tv_sec) + (T_merge.tv_usec - T_map.tv_usec) / 1000;





        //======================SORT(var 2 - сортировка расческой)======================
        comb_sort(m2, N / 2);
        hipEventSynchronize(syncEvent);  //Синхронизируем event

        gettimeofday(&T_sort, nullptr);
        delta_sort = 1000 * (T_sort.tv_sec - T_merge.tv_sec) + (T_sort.tv_usec - T_merge.tv_usec) / 1000;




        //======================REDUCE======================
        reduce(m2, N / 2);

        gettimeofday(&T_result, nullptr);
        delta_reduce = 1000 * (T_result.tv_sec - T_sort.tv_sec) + (T_result.tv_usec - T_sort.tv_usec) / 1000;



        cout << delta_gen << ";" << delta_map << ";" << delta_merge << ";"
        << delta_merge << ";" << delta_sort << ";" << delta_reduce << endl;

        hipEventSynchronize(syncEvent);  //Синхронизируем event
    }

    hipEventDestroy(syncEvent);

    hipFree(m1);
    hipFree(m2);
    hipFree(m2_copy);

    free(m1);
    free(m2);
    free(m2_copy);

    gettimeofday(&T2, nullptr); // запомнить текущее время T2
    delta_ms = 1000 * (T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec) / 1000;
    // printf("\nN=%d. Milliseconds passed: %ld\n", N, delta_ms); /* T2 - T1 */
    printf("%d;%ld\n", N, delta_ms); /* T2 - T1 */
    return 0;
}