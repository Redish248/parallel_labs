
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <sys/time.h>

using namespace std;

const int A = 936;

__global__ void map_m1(double* m1_v, int size) {
    //линейный индекс потока
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    //сколько один поток выполняет
    unsigned int offset = blockDim.x * gridDim.x;
    for (unsigned int i = id; i < size; i += offset) {
        m1_v[i] = cosh(m1_v[i]) + 1;
    }
}

__global__ void map_m2(double* m2_v, double* m2_copy_v, int size) {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int offset = blockDim.x * gridDim.x;
    for (unsigned int i = id; i < size; i += offset) {
        if (i == 0) {
            m2_v[i] = fabs((double) 1 / tan(m2_v[i]));
        } else {
            m2_v[i] = fabs((double) 1 / tan(m2_v[i] + m2_copy_v[i - 1]));
        }
    }
}

__global__ void merge(const double* m1_v, double* m2_v, int size) {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int offset = blockDim.x * gridDim.x;
    for (unsigned int i = id; i < size; i += offset) {
        m2_v[i] = (double) m1_v[i] / m2_v[i];
    }
}

/* comb_sort: function to find the new gap between the elements */
void comb_sort(double data[], int size) { //
    double factor = 1.2473309; // фактор уменьшения
    long step = size - 1; // шаг сортировки

    while (step >= 1) {
        for (int i = 0; i + step < size; i++) {
            if (data[i] > data[i + step]) {
                double tmp = data[i];
                data[i] = data[i + step];
                data[i + step] = tmp;
            }
        }
        step /= factor;
    }
}

// data - sorted array !
double reduce(double data[], int size) {
    double result = 0;

    int j = 0;
    while (j < size && data[j] == 0) {
        j++;
    }
    double min = data[j];

    for (int i = 0; i < size; i++) {
        if (((long) (data[i] / min) % 2) == 0) {
            result += sin(data[i]);
        }
    }

    return result;
}

int main(int argc, char *argv[]) {
    int N, K;
    struct timeval T1, T2;
    long delta_ms;

    if (argc < 2) {
        printf("Need to add size of array as input arguments\n");
        return -1;
    }

    N = atoi(argv[1]);
    if (argc >= 3) {
        K = atoi(argv[2]);
    } else K = 100;

    double *m1, *m2, *m2_copy;
    double *m1v, *m2v, *m2_copyv;

    m1 = (double *) malloc(N * sizeof(double));
    m2 = (double *) malloc(N / 2 * sizeof(double));
    m2_copy = (double *) malloc(N / 2 * sizeof(double));

    //выделение памяти на устройстве
    hipMalloc(&m1v, sizeof(double) * N);
    hipMalloc(&m2v, sizeof(double) * N / 2);
    hipMalloc(&m2_copyv, sizeof(double) * N / 2);

    gettimeofday(&T1, nullptr); //запомнить текущее время T1

    //для сихронизации потоков
    hipEvent_t syncEvent;
    hipEventCreate(&syncEvent);    //Создаем event
    hipEventRecord(syncEvent, nullptr);  //Записываем event

    //расчёт gridSize и blockSize для m1
    int gridSize, blockSize, minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, map_m1, 0, N);
    gridSize = (N + blockSize - 1) / blockSize;

    //расчёт gridSize и blockSize для m2
    int minGridSize2, blockSize2, gridSize2;
    hipOccupancyMaxPotentialBlockSize(&minGridSize2, &blockSize2, map_m2, 0, N / 2);
    gridSize2 = (N / 2  + blockSize2 - 1) / blockSize2;

    for (unsigned int ink = 0; ink < K; ink++) {

        //======================GENERATE======================
        unsigned int tmp1 = ink;
        unsigned int tmp2 = ink;
        //Заполнить массив исходных данных размером N
        for (int j = 0; j < N; j++) {
            double value = 1 + rand_r(&tmp1) % (A - 1);
            m1[j] = value;
        }

        for (int j = 0; j < N / 2; j++) {
            double value = A + rand_r(&tmp2) % (A * 10 - A);
            m2[j] = value;
            m2_copy[j] = value;
        }

        //копирование данных после инициализации
        hipMemcpy(m1v, m1, sizeof(double) * N, hipMemcpyHostToDevice);
        hipMemcpy(m2v, m2, sizeof(double) * N / 2, hipMemcpyHostToDevice);
        hipMemcpy(m2_copyv, m2_copy, sizeof(double) * N / 2, hipMemcpyHostToDevice);





        //======================MAP======================

        map_m1<<<gridSize, blockSize>>>(m1v, N);
        map_m2<<<gridSize2, blockSize2>>>(m2v, m2_copyv, N / 2);

        //Хендл event'а
        hipEventSynchronize(syncEvent);  //Синхронизируем event

        hipMemcpy(m1, m1v, sizeof(double) * N, hipMemcpyDeviceToHost);
        hipMemcpy(m2, m2v, sizeof(double) * N / 2, hipMemcpyDeviceToHost);




        //======================MERGE======================
        merge<<<gridSize2, blockSize2>>>(m1v, m2v, N / 2);

        hipEventSynchronize(syncEvent);  //Синхронизируем event
        hipMemcpy(m2, m2v, sizeof(double) * N / 2, hipMemcpyDeviceToHost);





        //======================SORT(var 2 - сортировка расческой)======================
        comb_sort(m2, N / 2);

        hipEventSynchronize(syncEvent);  //Синхронизируем event




        //======================REDUCE======================
        reduce(m2, N / 2);

        hipEventSynchronize(syncEvent);  //Синхронизируем event
    }

    hipEventDestroy(syncEvent);

    hipFree(m1);
    hipFree(m2);
    hipFree(m2_copy);

    free(m1);
    free(m2);
    free(m2_copy);

    gettimeofday(&T2, nullptr); // запомнить текущее время T2
    delta_ms = 1000 * (T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec) / 1000;
   // printf("\nN=%d. Milliseconds passed: %ld\n", N, delta_ms); /* T2 - T1 */
    printf("%d;%ld\n", N, delta_ms); /* T2 - T1 */
    return 0;
}