
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>

using namespace std;

const int A = 936;

__global__ void sum(double *a, double *b, double *c, int N) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int threadsNum = blockDim.x * gridDim.x;
    for (int i = id; i < N; i += threadsNum)
        c[i] = a[i] + b[i];
}

int main(int argc, char *argv[]) {
    int N, M, K;
    struct timeval T1, T2;
    long delta_ms;

    if (argc < 3) {
        printf("Need to add size of array and number of threads as input arguments\n");
        return -1;
    }

    N = atoi(argv[1]);
    M = atoi(argv[2]);
    if (argc >= 4) {
        K = atoi(argv[3]);
    } else K = 100;
    double *m1, *m2, *m2_copy;
    double *m1v, *m2v, *m2_copyv;

    m1 = (double *) malloc(N * sizeof(double));
    m2 = (double *) malloc(N / 2 * sizeof(double));
    m2_copy = (double *) malloc(N / 2 * sizeof(double));

    hipMalloc(&m1v, sizeof(double) * N);
    hipMalloc(&m2v, sizeof(double) * N / 2);
    hipMalloc(&m2_copyv, sizeof(double) * N / 2);

    gettimeofday(&T1, nullptr);

    // TODO: 100 экспериментов - вернуть цикл
    //GENERATE:
    unsigned int tmp1 = 10; //TODO: tmp1 = i;
    unsigned int tmp2 = 10; // tODO: tmp2 = i;
    //Заполнить массив исходных данных размером N
    for (int j = 0; j < N; j++) {
        double value = 1 + rand_r(&tmp1) % (A - 1);
        m1[j] = value;
    }

    for (int j = 0; j < N / 2; j++) {
        double value = A + rand_r(&tmp2) % (A * 10 - A);
        m2[j] = value;
       // m2_copy[j] = value;
    }

    for (int i = 0; i < N; i++) {
        cout << "m1 " << m1[i] << "\n";
    }

    for (int i = 0; i < N / 2; i++) {
        cout << "m2 " << m2[i] << "\n";
    }

    hipMemcpy(m1v, m1, sizeof(double) * N, hipMemcpyHostToDevice);
    hipMemcpy(m2v, m2, sizeof(double) * N / 2, hipMemcpyHostToDevice);
    hipMemcpy(m2_copyv, m2_copy, sizeof(double) * N / 2, hipMemcpyHostToDevice);

    dim3 gridSize = dim3(1, 1, 1);    //TODO: Размер используемого грида
    dim3 blockSize = dim3(N / 2, 1, 1); //TODO: Размер используемого блока

    sum<<<gridSize, blockSize>>>(m1v, m2v, m2_copyv, N / 2);

    //Хендл event'а
    hipEvent_t syncEvent;

    hipEventCreate(&syncEvent);    //Создаем event
    hipEventRecord(syncEvent, nullptr);  //Записываем event
    hipEventSynchronize(syncEvent);  //Синхронизируем event

    hipMemcpy(m2_copy, m2_copyv, sizeof(double) * N / 2, hipMemcpyDeviceToHost);

    for (int i = 0; i < N / 2; i++) {
        cout << m2_copy[i] << " ";
    }

    hipEventDestroy(syncEvent);

    hipFree(m1);
    hipFree(m2);
    hipFree(m2_copy);

    free(m1);
    free(m2);
    free(m2_copy);

    gettimeofday(&T2, nullptr); // запомнить текущее время T2
    delta_ms = 1000 * (T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec) / 1000;
   // printf("\nN=%d. Milliseconds passed: %ld\n", N, delta_ms); /* T2 - T1 */
    printf("%d;%ld\n", N, delta_ms); /* T2 - T1 */
    return 0;
}