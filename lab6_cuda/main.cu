
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <sys/time.h>

using namespace std;

const int A = 936;

__global__ void map_m1(double* m1_v, int size) {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int threadsNum = blockDim.x * gridDim.x;
    for (unsigned int i = id; i < size; i += threadsNum) {
        m1_v[i] = cosh(m1_v[i]) + 1;
    }
}

__global__ void map_m2(double* m2_v, double* m2_copy_v, int size) {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int threadsNum = blockDim.x * gridDim.x;
    for (unsigned int i = id; i < size; i += threadsNum) {
        if (i == 0) {
            m2_v[i] = fabs((double) 1 / tan(m2_v[i]));
        } else {
            m2_v[i] = fabs((double) 1 / tan(m2_v[i] + m2_copy_v[i - 1]));
        }
    }
}

__global__ void merge(const double* m1_v, double* m2_v, int size) {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int threadsNum = blockDim.x * gridDim.x;
    for (unsigned int i = id; i < size; i += threadsNum) {
        m2_v[i] = (double) m1_v[i] / m2_v[i];
    }
}

/* comb_sort: function to find the new gap between the elements */
void comb_sort(double data[], int size) { //
    double factor = 1.2473309; // фактор уменьшения
    long step = size - 1; // шаг сортировки

    while (step >= 1) {
        for (int i = 0; i + step < size; i++) {
            if (data[i] > data[i + step]) {
                double tmp = data[i];
                data[i] = data[i + step];
                data[i + step] = tmp;
            }
        }
        step /= factor;
    }
}

int main(int argc, char *argv[]) {
    int N, M, K;
    struct timeval T1, T2;
    long delta_ms;

    if (argc < 2) {
        printf("Need to add size of array as input arguments\n");
        return -1;
    }

    N = atoi(argv[1]);
    if (argc >= 4) {
        K = atoi(argv[2]);
    } else K = 100;

    double *m1, *m2, *m2_copy;
    double *m1v, *m2v, *m2_copyv;

    m1 = (double *) malloc(N * sizeof(double));
    m2 = (double *) malloc(N / 2 * sizeof(double));
    m2_copy = (double *) malloc(N / 2 * sizeof(double));

    //выделение памяти на устройстве
    hipMalloc(&m1v, sizeof(double) * N);
    hipMalloc(&m2v, sizeof(double) * N / 2);
    hipMalloc(&m2_copyv, sizeof(double) * N / 2);

    gettimeofday(&T1, nullptr); //запомнить текущее время T1

    //для сихронизации потоков
    hipEvent_t syncEvent;
    hipEventCreate(&syncEvent);    //Создаем event
    hipEventRecord(syncEvent, nullptr);  //Записываем event

    dim3 gridSize = dim3(1, 1, 1);    //TODO: Размер используемого грида -- нужен ли фикс?
    dim3 blockSize = dim3(N / 2, 1, 1); //TODO: Размер используемого блока -- нужен ли фикс?

    for (unsigned int ink = 0; ink < K; ink++) {

        //======================GENERATE======================
        unsigned int tmp1 = ink;
        unsigned int tmp2 = ink;
        //Заполнить массив исходных данных размером N
        for (int j = 0; j < N; j++) {
            double value = 1 + rand_r(&tmp1) % (A - 1);
            m1[j] = value;
        }

        for (int j = 0; j < N / 2; j++) {
            double value = A + rand_r(&tmp2) % (A * 10 - A);
            m2[j] = value;
            m2_copy[j] = value;
        }

       /* for (int i = 0; i < N; i++) {
            cout << "m1 " << m1[i] << "\n";
        }

        for (int i = 0; i < N / 2; i++) {
            cout << "m2 " << m2[i] << "\n";
        }
        cout << "\n";*/

        //копирование данных после инициализации
        hipMemcpy(m1v, m1, sizeof(double) * N, hipMemcpyHostToDevice);
        hipMemcpy(m2v, m2, sizeof(double) * N / 2, hipMemcpyHostToDevice);
        hipMemcpy(m2_copyv, m2_copy, sizeof(double) * N / 2, hipMemcpyHostToDevice);


        //======================MAP======================

        map_m1<<<gridSize, blockSize>>>(m1v, N);
        map_m2<<<gridSize, blockSize>>>(m2v, m2_copyv, N / 2);

        //Хендл event'а
        hipEventSynchronize(syncEvent);  //Синхронизируем event

        hipMemcpy(m1, m1v, sizeof(double) * N, hipMemcpyDeviceToHost);
        hipMemcpy(m2, m2v, sizeof(double) * N / 2, hipMemcpyDeviceToHost);

      /*  for (int i = 0; i < N; i++) {
            cout << "map m1 " << m1[i] << "\n";
        }

        for (int i = 0; i < N / 2; i++) {
            cout << "map m2 " << m2[i] << "\n";
        }
        cout << "\n";*/




        //======================MERGE======================
        merge<<<gridSize, blockSize>>>(m1v, m2v, N / 2);

        hipEventSynchronize(syncEvent);  //Синхронизируем event
        hipMemcpy(m2, m2v, sizeof(double) * N / 2, hipMemcpyDeviceToHost);

       /* for (int i = 0; i < N / 2; i++) {
            cout << "merge m2 " << m2[i] << "\n";
        }
        cout << "\n";*/



        //======================SORT(var 2 - сортировка расческой)======================
        comb_sort(m2, N / 2);

        /*for (int i = 0; i < N / 2; i++) {
            cout << "sort m2 " << m2[i] << "\n";
        }
        cout << "\n";*/

        hipEventSynchronize(syncEvent);  //Синхронизируем event


        //======================RECUCE======================
        double result = 0;
        int j = 0;
        while (j < N / 2 && m2[j] == 0) {
            j++;
        }
        double min = m2[j];

        for (int i = 0; i < N / 2; i++) {
            if (((long) (m2[i] / min) % 2) == 0) {
                result += sin(m2[i]);
            }
        }

        cout << ink << " X: " << result << "\n";

        hipEventSynchronize(syncEvent);  //Синхронизируем event
    }

    hipEventDestroy(syncEvent);

    hipFree(m1);
    hipFree(m2);
    hipFree(m2_copy);

    free(m1);
    free(m2);
    free(m2_copy);

    gettimeofday(&T2, nullptr); // запомнить текущее время T2
    delta_ms = 1000 * (T2.tv_sec - T1.tv_sec) + (T2.tv_usec - T1.tv_usec) / 1000;
   // printf("\nN=%d. Milliseconds passed: %ld\n", N, delta_ms); /* T2 - T1 */
    printf("%d;%ld\n", N, delta_ms); /* T2 - T1 */
    return 0;
}