
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <time.h>

using namespace std;

const int A = 936;

__global__ void sum(double *a, double *b, double *c, int N) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int threadsNum = blockDim.x * gridDim.x;
    for (int i = id; i < N; i += threadsNum)
        c[i] = a[i] + b[i];
}

int main(int argc, char *argv[]) {
    int N, M, K;
    struct timeval T1, T2;
    long delta_ms;

    if (argc < 3) {
        printf("Need to add size of array and number of threads as input arguments\n");
        return -1;
    }

    N = atoi(argv[1]);
    M = atoi(argv[2]);
    if (argc >= 4) {
        K = atoi(argv[3]);
    } else K = 100;
    double *m1, *m2, *m2_copy;
    double *m1v, *m2v, *m2_copyv;

    m1 = (double *) malloc(N * sizeof(double));
    m2 = (double *) malloc(N / 2 * sizeof(double));
    m2_copy = (double *) malloc(N / 2 * sizeof(double));

    hipMalloc(&m1v, sizeof(double) * N);
    hipMalloc(&m2v, sizeof(double) * N / 2);
    hipMalloc(&m2_copyv, sizeof(double) * N / 2);

    //gettimeofday(&T1, NULL)

    // 100 экспериментов
   /* for (unsigned int i = 0; i < K; i++) {
        //GENERATE:
        unsigned int tmp1 = i;
        unsigned int tmp2 = i;
        //Заполнить массив исходных данных размером N
        for (int j = 0; j < N; j++) {
            double value = 1 + rand_r(&tmp1) % (A - 1);
            m1[j] = value;
        }

        for (int j = 0; j < N / 2; j++) {
            double value = A + rand_r(&tmp2) % (A * 10 - A);
            m2[j] = value;
            m2_copy[j] = value;
        }


    }*/

   m1[0] = 1;
   m1[1] = 2;
   m1[2] = 3;
    m1[3] = 4;
    m1[4] = 5;
    m1[5] = 6;
    m2[0] = 1;
    m2[1] = 2;
    m2[2] = 3;

    hipMemcpy(m1v, m1, sizeof(double) * N, hipMemcpyHostToDevice);
    hipMemcpy(m2v, m2, sizeof(double) * N / 2, hipMemcpyHostToDevice);
    hipMemcpy(m2_copyv, m2_copy, sizeof(double) * N / 2, hipMemcpyHostToDevice);

    dim3 gridSize = dim3(1, 1, 1);    //Размер используемого грида
    dim3 blockSize = dim3(N / 2, 1, 1); //Размер используемого блока


    sum<<<gridSize, blockSize>>>(m1v, m2v, m2_copyv, N / 2 );

    //Хендл event'а
    hipEvent_t syncEvent;

    hipEventCreate(&syncEvent);    //Создаем event
    hipEventRecord(syncEvent, nullptr);  //Записываем event
    hipEventSynchronize(syncEvent);  //Синхронизируем event

    hipMemcpy(m2_copy, m2_copyv, sizeof(double) * N / 2, hipMemcpyDeviceToHost);

    for (int i = 0; i < N / 2; i++) {
        cout << m2_copy[i] << " ";
    }

    hipEventDestroy(syncEvent);


    hipFree(m1);
    hipFree(m2);
    hipFree(m2_copy);

    //gettimeofday(&T2, NULL); // запомнить текущее время T2
    //delta_ms = (T2 - T1) * 1000;
//    printf("\nN=%d. Milliseconds passed: %ld\n", N, delta_ms); /* T2 - T1 */
   // printf("%d;%ld\n", N, delta_ms); /* T2 - T1 */
    return 0;
}